#include "hip/hip_runtime.h"
#include "cov.hpp"
#include "cov_gpu.hpp"
#include <iostream>

// __device__ double cov_val_d(int n_dim, double *x, double *y, double *hypers)
// {
// 	double scale = hypers[0];
// 	//vec r(hypers.rows(1,hypers.n_rows-1));

// 	// bit of a hack.  Assumes a size on the vectors
// 	//vec r{hypers(1), hypers(1), hypers(1), hypers(2), hypers(2), hypers(2), hypers(3)};
// 	double r[] = {hypers[1], hypers[1], hypers[1], hypers[2], hypers[2], hypers[2], hypers[3]};

// 	double s = 0.0;
// 	for (unsigned i=0; i<n_dim; i++)
// 	{
// 		s+=pow(x[i]-y[i],2.0)/(r[i]*r[i]);
// 	}
// 	return scale * exp(-0.5*s);
// }


int main(void)
{
	const int Ninput = 7;

	vec x(Ninput); x.fill(0.0);
	vec y(Ninput); y.fill(0.1);
	vec hypers(4); hypers.fill(1.0);
	std::cout << cov(0, 0, x, y, hypers) << std::endl;
	
	double *x_d;
	double *y_d;
	double *hypers_d;
	double *result_d;

	if (hipMalloc((void**)(&x_d), Ninput*sizeof(double)) != hipSuccess) {
		throw std::runtime_error("Device allocation failure (x_d)");
	}
	if (hipMalloc((void**)(&y_d), Ninput*sizeof(double)) != hipSuccess) {
		throw std::runtime_error("Device allocation failure (y_d)");
	}
	if (hipMalloc((void**)(&hypers_d), hypers.n_rows*sizeof(double)) != hipSuccess) {
		throw std::runtime_error("Device allocation failure (hypers_d)");
	}
	if (hipMalloc((void**)(&result_d), sizeof(double)) != hipSuccess) {
		throw std::runtime_error("Device allocation failure (result_d)");
	}

	hipMemcpy(x_d, x.memptr(), Ninput*sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(y_d, y.memptr(), Ninput*sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(hypers_d, hypers.memptr(), hypers.n_rows*sizeof(double), hipMemcpyHostToDevice);
	cov_val_wrapper(result_d, Ninput, x_d, y_d, hypers_d);
	hipError_t err = hipGetLastError();
	if (err != hipSuccess) 
		printf("Error: %s\n", hipGetErrorString(err));

	double result;
	hipMemcpy(&result, result_d, sizeof(double), hipMemcpyDeviceToHost);
	std::cout << result << std::endl;
}

